#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/cuda/Sort.h>
#include <ATen/core/TensorBase.h>
#include <ATen/core/Array.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/cub.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/SortUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>

#include <limits>
#include <c10/core/DeviceArray.h>

namespace at { namespace native {

template <typename T>
static int minimum_grid_for_occupancy(T kernel, int max_block_size) {
  int minGridSize;
  int blockSize;
  C10_CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(
      &minGridSize,
      &blockSize,
      kernel,
      /*dynamicSMemSize=*/0,
      max_block_size));
  return minGridSize;
}

// For very small sorts, use bitonicSortKVInPlace which performs
// better because it can sort multiple arrays within the same block of
// threads, improving occupancy.
//
// TODO: cub in CUDA 11.6 has a WarpMergeSort primitive that could
// replace the bitonic sort here.
struct SmallBitonicSort {
  template <int A, typename K, typename V, typename IndexType>
  void sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {
    constexpr int sort_size = 32;
    constexpr int max_block_y = 16;
    constexpr int items_per_thread = 2;
    static_assert(sort_size % items_per_thread == 0, "");
    constexpr int block_x = sort_size / items_per_thread;

    TORCH_INTERNAL_ASSERT(keySliceSize <= sort_size);

    // Scale batch size down if the grid would be too small
    const auto min_grid = minimum_grid_for_occupancy(
        bitonicSortKVInPlace<
            A, -1, block_x, max_block_y,
            K, V, LTOp<K, true>, IndexType>,
        block_x * max_block_y);
    const auto max_batch = std::max(IndexType{1}, keySlices / min_grid);
    const int block_y = std::min(IndexType(max_block_y), max_batch);
    dim3 block(block_x, block_y);

    dim3 grid;
    const int grid_count = (keySlices + block_y - 1) / block_y;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(grid_count, grid),
                          "Too many slices to sort");
    const auto stream = at::cuda::getCurrentCUDAStream();

    if (descending) {
      bitonicSortKVInPlace<A, -1, block_x, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          GTOp<K, true>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      bitonicSortKVInPlace<A, -1, block_x, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          LTOp<K, true>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
};

// For medium sizes (32 < n <= 4096) use radixSortKVInplace for better
// performance than the bitonic sort kernel.
struct MediumRadixSort {

  template <int A, typename K, typename V, typename IndexType>
  void sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {

#define HANDLE_CASE(SIZE, ITEMS_PER_THREAD)         \
    fixed_size_sort<A, SIZE, ITEMS_PER_THREAD>(     \
        keyInfo,                                    \
        keySlices,                                  \
        keySliceSize,                               \
        keySliceStride,                             \
        valueInfo,                                  \
        valueSliceStride,                           \
        descending)

    int64_t ceilPowerOf2 = nextHighestPowerOf2(keySliceSize);
    TORCH_INTERNAL_ASSERT(ceilPowerOf2 <= 4096);
    switch (ceilPowerOf2) {
      case 4096:
        HANDLE_CASE(4096, 32);
        break;
      case 2048:
        HANDLE_CASE(2048, 32);
        break;
      case 1024:
      case 512:
      case 256:
        HANDLE_CASE(1024, 32);
        break;
      case 128:
      case 64:
        HANDLE_CASE(128, 4);
        break;
      case 32:
      case 16:
      case 8:
      case 4:
      case 2:
        HANDLE_CASE(32, 2);
        break;
      case 1:
        /* Nothing to do, data already sorted */
        break;
      default:
        TORCH_INTERNAL_ASSERT(false);
    }
#undef HANDLE_CASE

  }

  template <int A, int sort_size, int items_per_thread,
            typename K, typename V, typename IndexType>
  void fixed_size_sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {
    static_assert(sort_size % items_per_thread == 0, "");
    constexpr int block = sort_size / items_per_thread;
    dim3 grid;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(keySlices, grid),
                          "Too many slices to sort");

    const auto stream = at::cuda::getCurrentCUDAStream();
    radixSortKVInPlace<A, -1, block, items_per_thread>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          descending);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
};

template <typename Sorter>
void sortCommon(Sorter sorter, const TensorBase &key, const TensorBase &value,
                int dim, bool descending) {
  TORCH_CHECK(key.sizes() == value.sizes(),
              "Key tensor must have same size as value tensor");
  int dims = value.dim();
  TORCH_CHECK(dims <= MAX_DIMS, "value tensor has too many dimensions");
  // if key and value tensors have the same size, we do not need to check both

  ptrdiff_t inElements = key.numel();

  if (inElements == 0) {
    return;
  }

  int64_t keySliceSize = key.size(dim);
  ptrdiff_t keySlices = inElements / keySliceSize;

#define HANDLE_SORT_CASE(TYPE, A)                   \
  sorter.template sort<A>(                          \
      keyInfo,                                      \
      (TYPE) keySlices,                             \
      (TYPE) keySliceSize,                          \
      (TYPE) keyInfo.strides[collapseKeyDim],       \
      valueInfo,                                    \
      (TYPE) valueInfo.strides[collapseValueDim],   \
      descending)

  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, key.scalar_type(), "sortKeyValueInplace", [&]  {
    if (at::cuda::detail::canUse32BitIndexMath(key)) {
      at::cuda::detail::TensorInfo<scalar_t, unsigned int> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, unsigned int>(key);
      at::cuda::detail::TensorInfo<int64_t, unsigned int> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, unsigned int>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      if (keyInfo.isContiguous()) {
        HANDLE_SORT_CASE(unsigned int, -2);
      } else {
        switch (keyInfo.dims) {
          case 2:
            HANDLE_SORT_CASE(unsigned int, 2);
            break;
          default:
            HANDLE_SORT_CASE(unsigned int, -1);
            break;
        }
      }

    } else {
      at::cuda::detail::TensorInfo<scalar_t, uint64_t> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, uint64_t>(key);
      at::cuda::detail::TensorInfo<int64_t, uint64_t> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, uint64_t>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      // int64_t case is rare, just instantiate the generic version
      HANDLE_SORT_CASE(uint64_t, -1);
    }
  });
#undef HANDLE_SORT_CASE
}

void sortKeyValueInplace(
    const TensorBase& key,
    const TensorBase& value,
    int dim,
    bool descending,
    bool stable) {
  if (!stable && key.size(dim) <= 32) {
    // NOTE: Bitonic sort is unstable
    sortCommon(SmallBitonicSort{}, key, value, dim, descending);
  } else {
    sortCommon(MediumRadixSort{}, key, value, dim, descending);
  }
}

}}  // namespace at::native
